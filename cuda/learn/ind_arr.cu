#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void unique_gid_calc_2d(int * input){
    int tix = threadIdx.x;
    int tiy = threadIdx.y;
    int bx = blockDim.x;
    int by = blockDim.y;
    int block_offsety = bx*tiy;
    int col_offsetx = bx*by*blockIdx.x;
    int row_offsety = bx*by*gridDim.x*blockIdx.y;
    int gid = tix+block_offsety+col_offsetx+row_offsety;
    printf("gid: %d, value: %d\n",gid,input[gid]);
}

int main() {
    int size = 16;
    int byte_size = sizeof(int)*size;
    int h_data[] = {23,9,4,53,65,12,1,33,10,11,50,49,38,5,2,7};
    for (int i=0;i<size;i++){
        printf("%d ",h_data[i]);
    }
    printf("\n");

    int * d_data;
    hipMalloc((void**)&d_data,byte_size);
    hipMemcpy(d_data,h_data,byte_size,hipMemcpyHostToDevice);

    dim3 block(2,2);
    dim3 grid(2,2);
    unique_gid_calc_2d <<<grid,block>>> (d_data);
    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}