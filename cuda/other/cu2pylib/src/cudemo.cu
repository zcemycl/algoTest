#include "hip/hip_runtime.h"
#include "cudemo.cuh"
#include <cstdio>
#include "hip/hip_runtime.h"
#include ""

// define a helper function for checking CUDA errors. See this thread: https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define cudaErrchk(ans) { GPUAssert((ans), __FILE__, __LINE__); }
inline void GPUAssert(hipError_t code, const char *file, int line, bool abort=true){
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

namespace pycudemo {

    __global__ void add_d(int *a_d,int *b_d,int *c_d){
        if (threadIdx.x==0)
            *c_d = *a_d + *b_d;
    }

    __host__ int add(int a,int b){
        int byte_size = sizeof(int);
        int *a_d,*b_d,*c_d, result;

        cudaErrchk(hipMalloc(&a_d,byte_size));
        cudaErrchk(hipMalloc(&b_d,byte_size));
        cudaErrchk(hipMalloc(&c_d,byte_size));

        cudaErrchk(hipMemcpy(a_d, &a, byte_size, hipMemcpyHostToDevice));
        cudaErrchk(hipMemcpy(b_d, &b, byte_size, hipMemcpyHostToDevice));

        add_d<<<1,1>>>(a_d,b_d,c_d);
        cudaErrchk(hipMemcpy(&result,c_d,byte_size,hipMemcpyDeviceToHost));


        return result;
    }



}